
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vector_add(float *a, float *b, float *c, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

// Calculate number of blocks needed
static unsigned num_blocks(int num, int factor) {
    return (num + factor - 1) / factor;
}

int main(int argc, char** argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s N\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);

    float *h_a = (float*)malloc(N * sizeof(float));
    float *h_b = (float*)malloc(N * sizeof(float));
    float *h_c = (float*)malloc(N * sizeof(float));
    if (!h_a || !h_b || !h_c) {
        fprintf(stderr, "Host malloc failed\n");
        return 1;
    }

    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2.0f;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    const int threadsPerBlock = 256;
    int blocksNeeded = num_blocks(N, threadsPerBlock);
    vector_add<<<blocksNeeded, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (int i = 0; i < N; i++)
        printf("%.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);

    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}

