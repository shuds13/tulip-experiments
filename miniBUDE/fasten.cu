
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h> 
#include <sys/time.h>

#define ZERO 0.0f
#define QUARTER 0.25f
#define HALF 0.5f
#define ONE 1.0f
#define TWO 2.0f
#define FOUR 4.0f
#define CNSTNT 45.0f
#define HARDNESS 38.0f
#define PPWI 64
#define HBTYPE_F 70
#define HBTYPE_E 69
#define NPNPDIST 5.5f
#define NPPDIST 1.0f
#define FloatMax 3.40282347e+38f  // std::numeric_limits<float>::max()
// #define USE_SHARED 0  // SH Changed from empty definition to value 1 for evaluation in expressions

// Define a simple Atom structure
struct Atom {
  float x, y, z;
  int type;
};

// Define FFParams structure like the original
struct FFParams {
  int hbtype;
  float radius;
  float hphb;
  float elsc;
};

// SH fabsf fails in splendid
// LLVM ERROR: Code generator does not support intrinsic function 'llvm.nvvm.fabs.f'!
static __forceinline__ __device__ __host__
float fasten_fabsf(float x) {
    return x < 0.0f ? -x : x;
}

// A simplified kernel implementation that avoids sin/cos and complex memory patterns
__global__ void fasten_kernel(int natlig, int natpro,
                           const Atom* protein_molecule,
                           const Atom* ligand_molecule,
                           const float* transforms_0, const float* transforms_1, const float* transforms_2,
                           const float* transforms_3, const float* transforms_4, const float* transforms_5,
                           float* etotals, const FFParams* forcefield, int numTransforms, int ntypes) {
  // Get index of first transform
  int ix = blockIdx.x * blockDim.x * PPWI + threadIdx.x;
  
  // Have extra threads do the last member instead of returning
  ix = ix < numTransforms ? ix : numTransforms - PPWI;
  
  // Use shared memory for forcefield parameters
// #ifdef USE_SHARED
//   extern __shared__ FFParams forcefield[];
//   if (threadIdx.x < ntypes) {
//     forcefield[threadIdx.x] = global_forcefield[threadIdx.x];
//   }
//   __syncthreads(); // Ensure all threads have loaded the forcefield data
// #else
//   const FFParams* forcefield = global_forcefield;
// #endif
  
  // Compute transformation matrices for all poses in this work item
  float transform[PPWI][3][4]; // [pose][row][column]
  
  const size_t lsz = blockDim.x;
  for (int i = 0; i < PPWI; i++) {
    size_t index = ix + i * lsz;
    if (index >= numTransforms) continue;
    
    // Load transform values for this pose
    transform[i][0][0] = transforms_0[index]; // transform_0_x
    transform[i][0][1] = transforms_1[index]; // transform_0_y
    transform[i][0][2] = transforms_2[index]; // transform_0_z
    transform[i][0][3] = transforms_3[index]; // transform_0_w
    transform[i][1][0] = transforms_0[index]; // transform_1_x - reusing
    transform[i][1][1] = transforms_1[index]; // transform_1_y - reusing
    transform[i][1][2] = transforms_2[index]; // transform_1_z - reusing
    transform[i][1][3] = transforms_4[index]; // transform_1_w
    transform[i][2][0] = transforms_0[index]; // transform_2_x - reusing
    transform[i][2][1] = transforms_1[index]; // transform_2_y - reusing
    transform[i][2][2] = transforms_2[index]; // transform_2_z - reusing
    transform[i][2][3] = transforms_5[index]; // transform_2_w
  }
  
  // Initialize energy results for all poses in this work item
  float etot[PPWI];
  for (int i = 0; i < PPWI; i++) {
    etot[i] = ZERO;
  }
  
  // Loop over ligand atoms
  for (int il = 0; il < natlig; il++) {
    // Load ligand atom data
    const Atom l_atom = ligand_molecule[il];
    const FFParams l_params = forcefield[l_atom.type];
    
    // Determine ligand atom parameters
    const bool lhphb_ltz = l_params.hphb < ZERO;
    const bool lhphb_gtz = l_params.hphb > ZERO;
    
    // Apply transforms to ligand atom for all poses in this work item
    float lpos[PPWI][3]; // [pose][coordinate]
    for (int i = 0; i < PPWI; i++) {
      // Apply transform
      lpos[i][0] = transform[i][0][3] + l_atom.x * transform[i][0][0] + l_atom.y * transform[i][0][1] + l_atom.z * transform[i][0][2];
      lpos[i][1] = transform[i][1][3] + l_atom.x * transform[i][1][0] + l_atom.y * transform[i][1][1] + l_atom.z * transform[i][1][2];
      lpos[i][2] = transform[i][2][3] + l_atom.x * transform[i][2][0] + l_atom.y * transform[i][2][1] + l_atom.z * transform[i][2][2];
    }
    
    // Loop over protein atoms
    for (int ip = 0; ip < natpro; ip++) {
      // Load protein atom data
      const Atom p_atom = protein_molecule[ip];
      const FFParams p_params = forcefield[p_atom.type];
      
      // Precalculate atom-specific parameters
      const float radij = p_params.radius + l_params.radius;
      const float r_radij = ONE / radij;
      
      const float elcdst = (p_params.hbtype == HBTYPE_F && l_params.hbtype == HBTYPE_F) ? FOUR : TWO;
      const float elcdst1 = (p_params.hbtype == HBTYPE_F && l_params.hbtype == HBTYPE_F) ? QUARTER : HALF;
      const bool type_E = ((p_params.hbtype == HBTYPE_E || l_params.hbtype == HBTYPE_E));
      
      const bool phphb_ltz = p_params.hphb < ZERO;
      const bool phphb_gtz = p_params.hphb > ZERO;
      const bool phphb_nz = p_params.hphb != ZERO;
      const float p_hphb = p_params.hphb * (phphb_ltz && lhphb_gtz ? -ONE : ONE);
      const float l_hphb = l_params.hphb * (phphb_gtz && lhphb_ltz ? -ONE : ONE);
      
      const float distdslv = (phphb_ltz ? (lhphb_ltz ? NPNPDIST : NPPDIST) : (lhphb_ltz ? NPPDIST : -FloatMax));
      const float r_distdslv = ONE / distdslv;
      
      const float chrg_init = l_params.elsc * p_params.elsc;
      const float dslv_init = p_hphb + l_hphb;
      
      // Calculate energy contribution for each pose in this work item
      for (int i = 0; i < PPWI; i++) {
        // Calculate distance between atoms
        const float dx = lpos[i][0] - p_atom.x;
        const float dy = lpos[i][1] - p_atom.y;
        const float dz = lpos[i][2] - p_atom.z;
        const float dist2 = dx*dx + dy*dy + dz*dz;
        const float dist = sqrtf(dist2);
        
        // Calculate steric energy
        const float distbb = dist - radij;
        const bool zone1 = (distbb < ZERO);
        etot[i] += (ONE - (dist * r_radij)) * (zone1 ? TWO * HARDNESS : ZERO);
        
        // Calculate formal and dipole charge interactions
        float chrg_e = chrg_init;
        chrg_e *= ((zone1 ? ONE : (ONE - distbb * elcdst1)) * (distbb < elcdst ? ONE : ZERO));
        const float neg_chrg_e = -fasten_fabsf(chrg_e);
        chrg_e = type_E ? neg_chrg_e : chrg_e;
        etot[i] += chrg_e * CNSTNT;
        
        // Calculate the two cases for Nonpolar-Polar repulsive interactions
        float dslv_e = dslv_init * ((distbb < distdslv && phphb_nz) ? ONE : ZERO);
        dslv_e *= (zone1 ? ONE : (ONE - distbb * r_distdslv));
        etot[i] += dslv_e;
      }
    }
  }
  
  // Store results for all poses in this work item
  const int td_base = blockIdx.x * blockDim.x * PPWI + threadIdx.x;
  for (int i = 0; i < PPWI; i++) {
    const int out_index = td_base + i * blockDim.x;
    if (out_index < numTransforms) {
      etotals[out_index] = etot[i] * HALF;
    }
  }
}

// Function to get time in milliseconds
double getTimeMs() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (tv.tv_sec * 1000.0 + tv.tv_usec / 1000.0);
}

// Function to get formatted time string
void getTimeString(char *buf, size_t len) {
    time_t now = time(NULL);
    struct tm *timeinfo = localtime(&now);
    strftime(buf, len, "%Y-%m-%d %H:%M:%S", timeinfo);
}

// int main(int argc, char **argv)
int main(void)
{
    // Default parameters
    int iterations = 1;
    int numThreads = 64;
    
    // // Parse command line arguments
    // for (int i = 1; i < argc; i++) {
    //     if (strcmp(argv[i], "--iter") == 0 && i+1 < argc) {
    //         iterations = atoi(argv[++i]);
    //     } else if (strcmp(argv[i], "--wgsize") == 0 && i+1 < argc) {
    //         numThreads = atoi(argv[++i]);
    //     } else if (strcmp(argv[i], "--help") == 0 || strcmp(argv[i], "-h") == 0) {
    //         printf("Usage: %s [options]\n", argv[0]);
    //         printf("Options:\n");
    //         printf("  --iter N        Number of iterations (default: 1)\n");
    //         printf("  --wgsize N      Work group size (default: 64)\n");
    //         printf("  --help, -h      Show this help message\n");
    //         return 0;
    //     }
    // }
    
    // Display header with time
    char timeStr[64];
    getTimeString(timeStr, sizeof(timeStr));
    printf("=== FASTEN CUDA Implementation ===\n");
    printf("Time: %s\n\n", timeStr);
    
    // Initialize with sample data
    int natpro = 5;
    int natlig = 3;
    int ntypes = 10;
    int nposes = 32;
    
    printf("Using sample data: %d ligands, %d proteins, %d forcefield types, %d poses\n", 
          natlig, natpro, ntypes, nposes);
    
    // Allocate host memory
    Atom *h_protein = (Atom*)malloc(natpro * sizeof(Atom));
    Atom *h_ligand = (Atom*)malloc(natlig * sizeof(Atom));
    FFParams *h_forcefield = (FFParams*)malloc(ntypes * sizeof(FFParams));
    float *h_transforms_0 = (float*)malloc(nposes * sizeof(float));
    float *h_transforms_1 = (float*)malloc(nposes * sizeof(float));
    float *h_transforms_2 = (float*)malloc(nposes * sizeof(float));
    float *h_transforms_3 = (float*)malloc(nposes * sizeof(float));
    float *h_transforms_4 = (float*)malloc(nposes * sizeof(float));
    float *h_transforms_5 = (float*)malloc(nposes * sizeof(float));
    float *h_results = (float*)malloc(nposes * sizeof(float));
    
    // Initialize protein array
    for (int i = 0; i < natpro; i++) {
        h_protein[i].x = i * 0.1f;
        h_protein[i].y = i * 0.2f;
        h_protein[i].z = i * 0.3f;
        h_protein[i].type = i % ntypes;
    }
    
    // Initialize ligand array
    for (int i = 0; i < natlig; i++) {
        h_ligand[i].x = i * 0.5f;
        h_ligand[i].y = i * 0.6f;
        h_ligand[i].z = i * 0.7f;
        h_ligand[i].type = i % ntypes;
    }
    
    // Initialize forcefield parameters
    for (int i = 0; i < ntypes; i++) {
        h_forcefield[i].hbtype = (i % 2 == 0) ? HBTYPE_F : HBTYPE_E;
        h_forcefield[i].radius = 1.5f + i * 0.1f;
        h_forcefield[i].hphb = (i % 2 == 0) ? -1.0f : 1.0f; 
        h_forcefield[i].elsc = 0.5f + i * 0.05f;
    }
    
    // Initialize transform data - using angles directly
    for (int i = 0; i < nposes; i++) {
        h_transforms_0[i] = 0.1f * i;  // Angle X
        h_transforms_1[i] = 0.2f * i;  // Angle Y
        h_transforms_2[i] = 0.3f * i;  // Angle Z
        h_transforms_3[i] = 10.0f + i * 0.1f;  // Translation X
        h_transforms_4[i] = 5.0f - i * 0.05f;  // Translation Y
        h_transforms_5[i] = -3.0f + i * 0.02f; // Translation Z
    }
    
    // Allocate device memory - WITHOUT error checking
    Atom *d_protein;
    Atom *d_ligand;
    FFParams *d_forcefield;
    float *d_transforms_0, *d_transforms_1, *d_transforms_2;
    float *d_transforms_3, *d_transforms_4, *d_transforms_5;
    float *d_results;
    
    hipMalloc(&d_protein, natpro * sizeof(Atom));
    hipMalloc(&d_ligand, natlig * sizeof(Atom));
    hipMalloc(&d_forcefield, ntypes * sizeof(FFParams));
    hipMalloc(&d_transforms_0, nposes * sizeof(float));
    hipMalloc(&d_transforms_1, nposes * sizeof(float));
    hipMalloc(&d_transforms_2, nposes * sizeof(float));
    hipMalloc(&d_transforms_3, nposes * sizeof(float));
    hipMalloc(&d_transforms_4, nposes * sizeof(float));
    hipMalloc(&d_transforms_5, nposes * sizeof(float));
    hipMalloc(&d_results, nposes * sizeof(float));
    hipMalloc(&d_forcefield, ntypes * sizeof(FFParams));
    hipMemcpy(d_forcefield, h_forcefield, ntypes * sizeof(FFParams), hipMemcpyHostToDevice);

    // Copy host memory to device - WITHOUT error checking
    hipMemcpy(d_protein, h_protein, natpro * sizeof(Atom), hipMemcpyHostToDevice);
    hipMemcpy(d_ligand, h_ligand, natlig * sizeof(Atom), hipMemcpyHostToDevice);
    hipMemcpy(d_forcefield, h_forcefield, ntypes * sizeof(FFParams), hipMemcpyHostToDevice);
    hipMemcpy(d_transforms_0, h_transforms_0, nposes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transforms_1, h_transforms_1, nposes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transforms_2, h_transforms_2, nposes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transforms_3, h_transforms_3, nposes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transforms_4, h_transforms_4, nposes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_transforms_5, h_transforms_5, nposes * sizeof(float), hipMemcpyHostToDevice);
    
    // Calculate grid and block dimensions
    int threadsPerBlock = numThreads;
    int blocksPerGrid = (nposes + threadsPerBlock * PPWI - 1) / (threadsPerBlock * PPWI);
    
    printf("\nRunning FASTEN kernel with %d blocks of %d threads for %d poses\n", 
           blocksPerGrid, threadsPerBlock, nposes);
    printf("Settings: %d iterations, PPWI=%d\n", iterations, PPWI);
    
    // Start timing
    double startTime = getTimeMs();
    
    // Calculate shared memory size for forcefield parameters
    size_t sharedMemSize = ntypes * sizeof(FFParams);
    
    // Run for multiple iterations if specified
    for (int iter = 0; iter < iterations; iter++) {
        // Execute kernel with multiple threads/blocks and shared memory
        fasten_kernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(
            natlig, natpro, d_protein, d_ligand,
            d_transforms_0, d_transforms_1, d_transforms_2,
            d_transforms_3, d_transforms_4, d_transforms_5,
            d_results, d_forcefield, nposes, ntypes
        );
        
        // Simple synchronization - without error checking
        hipDeviceSynchronize();
    }
    
    // End timing
    double endTime = getTimeMs();
    double totalTime = endTime - startTime;
    double timePerIteration = totalTime / iterations;
    
    // Copy device memory to host - WITHOUT error checking
    hipMemcpy(h_results, d_results, nposes * sizeof(float), hipMemcpyDeviceToHost);
    
    // Calculate statistics
    float minEnergy = h_results[0];
    float maxEnergy = h_results[0];
    float avgEnergy = 0.0f;
    
    for (int i = 0; i < nposes; i++) {
        if (h_results[i] < minEnergy) minEnergy = h_results[i];
        if (h_results[i] > maxEnergy) maxEnergy = h_results[i];
        avgEnergy += h_results[i];
    }
    avgEnergy /= nposes;
    
    // // Print timing and performance information
    // printf("\nPerformance Results:\n");
    // printf("  Total Time: %.3f ms\n", totalTime);
    // printf("  Time per Iteration: %.3f ms\n", timePerIteration);
    // printf("  Poses Processed: %d\n", nposes * iterations);
    // printf("  Processing Rate: %.2f poses/ms (%.2f poses/second)\n",
    //        (nposes * iterations) / totalTime,
    //        (nposes * iterations) / totalTime * 1000.0);
    
    // // Print energy statistics
    // printf("\nEnergy Statistics:\n");
    // printf("  Min Energy: %.4f\n", minEnergy);
    // printf("  Max Energy: %.4f\n", maxEnergy);
    // printf("  Avg Energy: %.4f\n", avgEnergy);
    
    // // Print sample energy values
    // printf("\nEnergy results (showing %d):\n", nposes < 10 ? nposes : 10);
    // for (int i = 0; i < nposes && i < 10; i++) {
    //     printf("  Pose %d: %.4f\n", i, h_results[i]);
    // }
    
    // Free memory
    free(h_protein);
    free(h_ligand);
    free(h_forcefield);
    free(h_transforms_0);
    free(h_transforms_1);
    free(h_transforms_2);
    free(h_transforms_3);
    free(h_transforms_4);
    free(h_transforms_5);
    free(h_results);
    
    hipFree(d_protein);
    hipFree(d_ligand);
    hipFree(d_forcefield);
    hipFree(d_transforms_0);
    hipFree(d_transforms_1);
    hipFree(d_transforms_2);
    hipFree(d_transforms_3);
    hipFree(d_transforms_4);
    hipFree(d_transforms_5);
    hipFree(d_results);
    
    return 0;
}
