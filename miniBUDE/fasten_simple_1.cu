
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Define a simple Atom structure
struct Atom {
    float x, y, z;
    int type;
};

#define N 10

__global__ void vector_add(float *a, float *b, float *c, int n)
{
    int i = threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

// A simple kernel that uses Atom structure but has same structure as vector_add
__global__ void atom_add(Atom *atoms_a, Atom *atoms_b, Atom *atoms_c, int n)
{
    int i = threadIdx.x;
    if (i < n) {
        atoms_c[i].x = atoms_a[i].x + atoms_b[i].x;
        atoms_c[i].y = atoms_a[i].y + atoms_b[i].y;
        atoms_c[i].z = atoms_a[i].z + atoms_b[i].z;
        atoms_c[i].type = atoms_a[i].type;
    }
}

int main()
{
    // Test vector_add first to verify it still works
    // Allocate host memory
    float *h_a = (float*)malloc(N * sizeof(float));
    float *h_b = (float*)malloc(N * sizeof(float));
    float *h_c = (float*)malloc(N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
        h_b[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy host memory to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Execute kernel
    vector_add<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy device memory to host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify results
    printf("Vector addition results:\n");
    for (int i = 0; i < 3; i++) // Only print first few results
    {
        printf("%.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Now test atom_add
    // Allocate host memory for Atoms
    Atom *h_atoms_a = (Atom*)malloc(N * sizeof(Atom));
    Atom *h_atoms_b = (Atom*)malloc(N * sizeof(Atom));
    Atom *h_atoms_c = (Atom*)malloc(N * sizeof(Atom));

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_atoms_a[i].x = i * 0.1f;
        h_atoms_a[i].y = i * 0.2f;
        h_atoms_a[i].z = i * 0.3f;
        h_atoms_a[i].type = i;
        
        h_atoms_b[i].x = i * 0.5f;
        h_atoms_b[i].y = i * 0.6f;
        h_atoms_b[i].z = i * 0.7f;
        h_atoms_b[i].type = i;
    }

    // Allocate device memory
    Atom *d_atoms_a, *d_atoms_b, *d_atoms_c;
    hipMalloc(&d_atoms_a, N * sizeof(Atom));
    hipMalloc(&d_atoms_b, N * sizeof(Atom));
    hipMalloc(&d_atoms_c, N * sizeof(Atom));

    // Copy host memory to device
    hipMemcpy(d_atoms_a, h_atoms_a, N * sizeof(Atom), hipMemcpyHostToDevice);
    hipMemcpy(d_atoms_b, h_atoms_b, N * sizeof(Atom), hipMemcpyHostToDevice);

    // Execute kernel
    atom_add<<<1, N>>>(d_atoms_a, d_atoms_b, d_atoms_c, N);

    // Copy device memory to host
    hipMemcpy(h_atoms_c, d_atoms_c, N * sizeof(Atom), hipMemcpyDeviceToHost);

    // Verify results
    printf("\nAtom addition results:\n");
    for (int i = 0; i < 3; i++) // Only print first few results
    {
        printf("Atom %d: (%.1f,%.1f,%.1f) + (%.1f,%.1f,%.1f) = (%.1f,%.1f,%.1f)\n", 
               i,
               h_atoms_a[i].x, h_atoms_a[i].y, h_atoms_a[i].z,
               h_atoms_b[i].x, h_atoms_b[i].y, h_atoms_b[i].z,
               h_atoms_c[i].x, h_atoms_c[i].y, h_atoms_c[i].z);
    }

    // Free memory
    free(h_atoms_a);
    free(h_atoms_b);
    free(h_atoms_c);
    hipFree(d_atoms_a);
    hipFree(d_atoms_b);
    hipFree(d_atoms_c);

    return 0;
}
