
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Define a simple Atom structure
struct Atom {
    float x, y, z;
    int type;
};

// Calculate number of blocks needed
static unsigned num_blocks(int num, int factor) {
    return (num + factor - 1) / factor;
}

__global__ void vector_add(float *a, float *b, float *c, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

// A simple kernel that uses Atom structure but has same structure as vector_add
__global__ void atom_add(Atom *atoms_a, Atom *atoms_b, Atom *atoms_c, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        atoms_c[i].x = atoms_a[i].x + atoms_b[i].x;
        atoms_c[i].y = atoms_a[i].y + atoms_b[i].y;
        atoms_c[i].z = atoms_a[i].z + atoms_b[i].z;
        atoms_c[i].type = atoms_a[i].type;
    }
}

int main(int argc, char** argv)
{
    // Check for command line argument
    if (argc < 2) {
        fprintf(stderr, "Usage: %s N\n", argv[0]);
        return 1;
    }
    int N = atoi(argv[1]);
    // Test vector_add first to verify it still works
    // Allocate host memory
    float *h_a = (float*)malloc(N * sizeof(float));
    float *h_b = (float*)malloc(N * sizeof(float));
    float *h_c = (float*)malloc(N * sizeof(float));

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
        h_b[i] = i * 2.0f;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy host memory to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Execute kernel
    // vector_add<<<1, N>>>(d_a, d_b, d_c, N);


    // Execute kernel
    const int threadsPerBlock = 256;
    int blocksNeeded = num_blocks(N, threadsPerBlock);
    vector_add<<<blocksNeeded, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();


    // Copy device memory to host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify results
    printf("Vector addition - first few results:\n");
    for (int i = 0; i < 10; i++) // Only print first few results
    {
        printf("%.1f + %.1f = %.1f\n", h_a[i], h_b[i], h_c[i]);
    }

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Now test atom_add
    // Allocate host memory for Atoms
    Atom *h_atoms_a = (Atom*)malloc(N * sizeof(Atom));
    Atom *h_atoms_b = (Atom*)malloc(N * sizeof(Atom));
    Atom *h_atoms_c = (Atom*)malloc(N * sizeof(Atom));

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_atoms_a[i].x = i * 0.1f;
        h_atoms_a[i].y = i * 0.2f;
        h_atoms_a[i].z = i * 0.3f;
        h_atoms_a[i].type = i;
        
        h_atoms_b[i].x = i * 0.5f;
        h_atoms_b[i].y = i * 0.6f;
        h_atoms_b[i].z = i * 0.7f;
        h_atoms_b[i].type = i;
    }

    // Allocate device memory
    Atom *d_atoms_a, *d_atoms_b, *d_atoms_c;
    hipMalloc(&d_atoms_a, N * sizeof(Atom));
    hipMalloc(&d_atoms_b, N * sizeof(Atom));
    hipMalloc(&d_atoms_c, N * sizeof(Atom));

    // Copy host memory to device
    hipMemcpy(d_atoms_a, h_atoms_a, N * sizeof(Atom), hipMemcpyHostToDevice);
    hipMemcpy(d_atoms_b, h_atoms_b, N * sizeof(Atom), hipMemcpyHostToDevice);

    // Execute kernel
    atom_add<<<blocksNeeded, threadsPerBlock>>>(d_atoms_a, d_atoms_b, d_atoms_c, N);
    hipDeviceSynchronize();

    // Copy device memory to host
    hipMemcpy(h_atoms_c, d_atoms_c, N * sizeof(Atom), hipMemcpyDeviceToHost);

    // Verify results
    printf("\nAtom addition - first few results:\n");
    for (int i = 0; i < 10; i++) // Only print first few results
    {
        printf("Atom %d: (%.1f,%.1f,%.1f) + (%.1f,%.1f,%.1f) = (%.1f,%.1f,%.1f)\n", 
               i,
               h_atoms_a[i].x, h_atoms_a[i].y, h_atoms_a[i].z,
               h_atoms_b[i].x, h_atoms_b[i].y, h_atoms_b[i].z,
               h_atoms_c[i].x, h_atoms_c[i].y, h_atoms_c[i].z);
    }

    // Free memory
    free(h_atoms_a);
    free(h_atoms_b);
    free(h_atoms_c);
    hipFree(d_atoms_a);
    hipFree(d_atoms_b);
    hipFree(d_atoms_c);

    return 0;
}
